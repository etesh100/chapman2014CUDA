#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void mykernel(){
	printf("Hello world from device!\n");
} /* end kernel */

int main(void) 
{
        /* launch this kernel 30 times*/
	mykernel<<< 30 , 1>>>();
 	hipDeviceSynchronize();
	printf("Hello World from Host\n");
	return 0;
} /* end main */
